
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void kernel(double *init_vct, double *res_vct, int n) {
    int ind = blockDim.x * blockIdx.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for(int i = ind; i < n; i += step) {
		res_vct[i] = init_vct[n - i - 1];
    }
}

void random(double *vct, int n) {
    double rand_state = 7.0;

    for(int i = 0; i < n; ++i) {
        vct[i] = ((double)rand() / (double)(RAND_MAX)) * rand_state;
    }
}

void scan(double *vct, int n) {
    double val = 0.0;
    for(int i = 0; i < n; ++i) {
        scanf("%lf", &val);
        vct[i] = val;
    }
}

void init(double *h_init, int n, bool random_flag) {
    if(random_flag)
        random(h_init, n);
    else
        scan(h_init, n);
}

int main() {
    bool random_flag = false;
    int n = 0;

    if(!random_flag)
        scanf("%d", &n);

    double *h_init = (double*)malloc(sizeof(double) * n);
    double *h_res = (double*)malloc(sizeof(double) * n);

    init(h_init, n, random_flag);

    double *d_init, *d_res;

    hipMalloc(&d_init, sizeof(double) * n);
    hipMemcpy(d_init, h_init, sizeof(double) * n, hipMemcpyHostToDevice);

    hipMalloc(&d_res, sizeof(double) * n);
    hipMemcpy(d_res, h_res, sizeof(double) * n, hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;

    float time = 0.0;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    kernel<<<32, 32>>>(d_init, d_res, n);
    hipEventRecord(stop, 0);

    hipEventSynchronize(start);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);

    hipEventDestroy(stop);
    hipEventDestroy(start);

    hipMemcpy(h_res, d_res, sizeof(double) * n, hipMemcpyDeviceToHost);

    if(!random_flag) {
        for(int i = 0; i < n; ++i)
            printf("%lf ", h_res[i]);
        printf("\n");
    }

    fprintf(stderr, "time = %f\n", time);

    hipFree(d_init);
    hipFree(d_res);

    free(h_init);
    free(h_res);

    return 0;
}
