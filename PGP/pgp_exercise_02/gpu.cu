#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define CSC(call)                                                     \
do {                                                                  \
    hipError_t res = call;                                           \
    if(res != hipSuccess) {                                          \
        fprintf(stderr, "ERROR in %s:%d. Message: %s\n",              \
                        __FILE__, __LINE__, hipGetErrorString(res)); \
        exit(0);                                                      \
    }                                                                 \
} while(0)

texture<uchar4, 2, hipReadModeElementType> tex;

__device__ int median(int *hist, int cnt) {
    int cumsum = 0;

    for(int i = 0; i < 256; ++i) {
        cumsum += hist[i];
        if(cumsum > cnt / 2) {
            return i;
        }
    }
    return 255;
}

__global__ void kernel(uchar4 *out, int w, int h, int r)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;

    uchar4 p;

    int rhist[256], ghist[256], bhist[256];
    int red, green, blue, cnt, x_start, x_end, y_start, y_end;

    for(int y = idy; y < h; y += offsety) {
        for(int x = idx; x < w; x += offsetx) {
            cnt = 0;

            for(int i = 0; i < 256; ++i) {
                rhist[i] = 0;
                ghist[i] = 0;
                bhist[i] = 0;
            }

            x_start = (x - r >= 0) ? x - r : 0;
            x_end = (x + r < w) ? x + r : w - 1;
            y_start = (y - r >= 0) ? y - r : 0;
            y_end = (y + r < h) ? y + r : h - 1;

            for(int m = y_start; m <= y_end; ++m) {
                for(int k = x_start; k <= x_end; ++k) {
                    p = tex2D(tex, k, m);
                    rhist[p.x] += 1;
                    ghist[p.y] += 1;
                    bhist[p.z] += 1;
                    cnt += 1;
                }
            }

            red = median(rhist, cnt);
            green = median(ghist, cnt);
            blue = median(bhist, cnt);

            out[x + y * w] = tex2D(tex, x, y);
            out[x + y * w].x = red;
            out[x + y * w].y = green;
            out[x + y * w].z = blue;
        }
    }
}


int main() {
    int w, h, r;
    char inp_filename[256], out_filename[256];
    
    scanf("%s", inp_filename);
    scanf("%s", out_filename);
    scanf("%d", &r);

    FILE *fp = fopen(inp_filename, "rb");
    fread(&w, sizeof(int), 1, fp);
    fread(&h, sizeof(int), 1, fp);
    uchar4 *data = (uchar4 *)malloc(sizeof(uchar4) * w * h);
    fread(data, sizeof(uchar4), w * h, fp);
    fclose(fp);

    hipArray *arr;
    hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
    CSC(hipMallocArray(&arr, &ch, w, h));
    CSC(hipMemcpyToArray(arr, 0, 0, data, sizeof(uchar4) * w * h, hipMemcpyHostToDevice));

    tex.normalized = false;
    tex.filterMode = hipFilterModePoint;
    tex.channelDesc = ch;
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;

    CSC(hipBindTextureToArray(tex, arr, ch));

    uchar4 *d_out;
    CSC(hipMalloc(&d_out, sizeof(uchar4) * w * h));

    hipEvent_t start;
    hipEvent_t stop;

    float time = 0.0;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    kernel<<<dim3(16, 16), dim3(32, 8)>>>(d_out, w, h, r);
    hipEventRecord(stop, 0);

    hipEventSynchronize(start);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);

    hipEventDestroy(stop);
    hipEventDestroy(start);

    CSC(hipGetLastError());
    CSC(hipMemcpy(data, d_out, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));
    CSC(hipUnbindTexture(tex));

    CSC(hipFreeArray(arr));
    CSC(hipFree(d_out));

    fp = fopen(out_filename, "wb");

    fwrite(&w, sizeof(int), 1, fp);
    fwrite(&h, sizeof(int), 1, fp);
    fwrite(data, sizeof(uchar4), w * h, fp);
    fclose(fp);

    free(data);
    fprintf(stderr, "time = %f\n", time);
    return 0;
}
